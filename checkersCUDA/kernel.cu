#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <SFML/Graphics.hpp>
#include <SFML/Window.hpp>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <hiprand/hiprand_kernel.h>

#define WINDOW_WIDTH 800
#define WINDOW_HEIGHT 800
#define BOARD_SIZE 8
#define PAWN_ROWS 3
#define PAWN_SIZE 80

#define MAX_MOVES 50


#define QUEEN_VALUE 80
#define PAWN_VALUE 30
#define PIECE_ROW_ADV 1
#define PIECE_MIDDLE_CENTER 4
#define PIECE_MIDDLE_SIDE -2
#define PIECE_CENTER_GOALIES 8
#define PIECE_SIDE_GOALIES 8
#define PIECE_DOUBLE_CORNER 4

#define PLAYER_VS_AI 0
#define PLAYER_ONE 1
#define PLAYER_TWO 2

// this should be multiply of 1024 otherwise it will get ceiled up to nearest multiplication of 1024
#define NUM_OF_EVAL_ONE 10240
#define TREE_ITER_ONE 50
#define PARALLEL_PLAYER_ONE false

// this should be multiply of 1024 otherwise it will get ceiled up to nearest multiplication of 1024
#define NUM_OF_EVAL_TWO 102400
#define TREE_ITER_TWO 50
#define PARALLEL_PLAYER_TWO true

#define MAX_BLOCK 1024

#define BLOCK_SIZE_ONE (NUM_OF_EVAL_ONE < MAX_BLOCK ? NUM_OF_EVAL_ONE : MAX_BLOCK)
#define BLOCK_SIZE_TWO (NUM_OF_EVAL_TWO < MAX_BLOCK ? NUM_OF_EVAL_TWO : MAX_BLOCK)


#define BLOCK_SIZE_ONE_V dim3((NUM_OF_EVAL_ONE < MAX_BLOCK ? NUM_OF_EVAL_ONE : MAX_BLOCK), 1, 1)
#define BLOCK_SIZE_TWO_V dim3((NUM_OF_EVAL_TWO < MAX_BLOCK ? NUM_OF_EVAL_TWO : MAX_BLOCK), 1, 1)


#define BLOCK_NUM_ONE (NUM_OF_EVAL_ONE / (float)BLOCK_SIZE_ONE != NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE ? NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE + 1 : NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE)
#define BLOCK_NUM_TWO (NUM_OF_EVAL_TWO / (float)BLOCK_SIZE_TWO != NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO ? NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO + 1 : NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO)

#define BLOCK_NUM_ONE_V dim3((NUM_OF_EVAL_ONE / (float)BLOCK_SIZE_ONE != NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE ? NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE + 1 : NUM_OF_EVAL_ONE / BLOCK_SIZE_ONE), 1, 1)
#define BLOCK_NUM_TWO_V dim3((NUM_OF_EVAL_TWO / (float)BLOCK_SIZE_TWO != NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO ? NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO + 1 : NUM_OF_EVAL_TWO / BLOCK_SIZE_TWO), 1, 1)



using namespace sf;
using namespace std;

typedef struct node {
    int* rows;
    int* cols;
    int* fields;
    bool* isQueen;
    bool blackTurn;
    int lastKill;
    int childSize;
    node** childs;
    node* parent;
    float avgReward;
    int howManyVisits;
} node;

typedef struct fixedNode {
    int rows[PAWN_ROWS * BOARD_SIZE];
    int cols[PAWN_ROWS * BOARD_SIZE];
    int fields[BOARD_SIZE * BOARD_SIZE];
    bool isQueen[PAWN_ROWS * BOARD_SIZE];
} fixedNode;

__host__ void recolorFields(RectangleShape* fields)
{
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        bool isBlack = ((i / BOARD_SIZE) + (i % BOARD_SIZE)) % 2 == 0;
        if (isBlack)
            fields[i].setFillColor(Color::Black);
    }
}

__host__ void setupFields(RectangleShape* fieldShapes, int* fields)
{
    const Vector2f vecSize{ (float)(WINDOW_WIDTH / BOARD_SIZE), (float)(WINDOW_HEIGHT / BOARD_SIZE) };
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        fields[i] = -1;
        fieldShapes[i].setSize(vecSize);

        const Vector2f vecPos{ (float)((i % BOARD_SIZE) * WINDOW_WIDTH / BOARD_SIZE),
            (float)(((BOARD_SIZE * BOARD_SIZE - 1 - i) / BOARD_SIZE) * WINDOW_HEIGHT / BOARD_SIZE) };
        fieldShapes[i].setPosition(vecPos);
    }
    recolorFields(fieldShapes);
}

__host__ void setPawnPosition(CircleShape& pawn, int row, int col)
{
    const Vector2f vecPos{ (float)(col * WINDOW_WIDTH / BOARD_SIZE + (WINDOW_WIDTH / BOARD_SIZE - PAWN_SIZE) / 2),
        (float)((BOARD_SIZE - 1 - row) * WINDOW_HEIGHT / BOARD_SIZE + (WINDOW_HEIGHT / BOARD_SIZE - PAWN_SIZE) / 2) };
    pawn.setPosition(vecPos);
}

__host__ void setupPawns(CircleShape* pawns, int* fields, int* rows, int* cols, bool* pawnHasKill, bool* isQueen)
{
    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
    {
        pawns[i].setRadius(PAWN_SIZE / 2);
        pawns[i].setOutlineThickness(3);
        pawns[i].setOutlineColor(Color::Red);

        int row, col;
        if (i < PAWN_ROWS * BOARD_SIZE / 2)
        {
            pawns[i].setFillColor(Color::White);
            row = i / (BOARD_SIZE / 2);
            col = 2 * (i % (BOARD_SIZE / 2)) + ((i / (BOARD_SIZE / 2)) % 2);
            setPawnPosition(pawns[i], row, col);

        }
        else
        {
            row = BOARD_SIZE - 1 - ((i % (PAWN_ROWS * BOARD_SIZE / 2)) / (BOARD_SIZE / 2));
            col = 2 * (i % (BOARD_SIZE / 2)) + ((i / (BOARD_SIZE / 2)) % 2);
            pawns[i].setFillColor(Color::Black);
            setPawnPosition(pawns[i], row, col);
        }
        fields[row * BOARD_SIZE + col] = i;
        rows[i] = row;
        cols[i] = col;
        pawnHasKill[i] = false;
        isQueen[i] = false;
    }
}

__host__ bool isClickInShape(Shape& shape, Vector2f clickPos)
{
    Vector2f shapePosition = shape.getPosition();
    return clickPos.x >= shapePosition.x && clickPos.x <= shapePosition.x + shape.getLocalBounds().width
        && clickPos.y >= shapePosition.y && clickPos.y <= shapePosition.y + shape.getLocalBounds().height;
}

__host__ __device__ void clearAvailableFields(bool* available, int& numOfAvailable)
{
    numOfAvailable = 0;
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
        available[i] = false;
}

__host__ __device__ bool hasQueenKill(int* fields, int row, int col, int idx)
{
    int halfPawn = PAWN_ROWS * BOARD_SIZE / 2;

    for (int r = row + 1, c = col - 1; r < BOARD_SIZE - 1 && c > 0; r++, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c - 1] < 0)
            {
                return true;
            }
            break;
        }
    }
    for (int r = row + 1, c = col + 1; r < BOARD_SIZE - 1 && c < BOARD_SIZE - 1; r++, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c + 1] < 0)
            {
                return true;
            }
            break;
        }
    }
    for (int r = row - 1, c = col - 1; r > 0 && c > 0; r--, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c - 1] < 0)
            {
                return true;
            }
            break;
        }
    }
    for (int r = row - 1, c = col + 1; r > 0 && c < BOARD_SIZE - 1; r--, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c + 1] < 0)
            {
                return true;
            }
            break;
        }
    }
    return false;

}

__host__ __device__ bool hasKill(int* fields, int idx, int* rows, int* cols, bool isChainKill = false)
{
    int halfPawns = PAWN_ROWS * BOARD_SIZE / 2;
    // white
    if (idx < PAWN_ROWS * BOARD_SIZE / 2 || isChainKill)
    {
        if (rows[idx] >= 0 && rows[idx] < BOARD_SIZE - 2)
        {
            if (cols[idx] > 1)
            {
                if (fields[(rows[idx] + 1) * BOARD_SIZE + cols[idx] - 1] >= 0 &&
                    fields[(rows[idx] + 1) * BOARD_SIZE + cols[idx] - 1] / halfPawns != idx / halfPawns &&
                    fields[(rows[idx] + 2) * BOARD_SIZE + cols[idx] - 2] < 0)
                    return true;
            }
            if (cols[idx] >= 0 && cols[idx] < BOARD_SIZE - 2)
            {
                if (fields[(rows[idx] + 1) * BOARD_SIZE + cols[idx] + 1] >= 0 &&
                    fields[(rows[idx] + 1) * BOARD_SIZE + cols[idx] + 1] / halfPawns != idx / halfPawns &&
                    fields[(rows[idx] + 2) * BOARD_SIZE + cols[idx] + 2] < 0)
                    return true;
            }
        }
    }
    // black
    if (idx >= PAWN_ROWS * BOARD_SIZE / 2 || isChainKill)
    {
        if (rows[idx] > 1)
        {
            if (cols[idx] > 1)
            {
                if (fields[(rows[idx] - 1) * BOARD_SIZE + cols[idx] - 1] >= 0 &&
                    fields[(rows[idx] - 1) * BOARD_SIZE + cols[idx] - 1] / halfPawns != idx / halfPawns &&
                    fields[(rows[idx] - 2) * BOARD_SIZE + cols[idx] - 2] < 0)
                    return true;
            }
            if (cols[idx] >= 0 && cols[idx] < BOARD_SIZE - 2)
            {
                if (fields[(rows[idx] - 1) * BOARD_SIZE + cols[idx] + 1] >= 0 &&
                    fields[(rows[idx] - 1) * BOARD_SIZE + cols[idx] + 1] / halfPawns != idx / halfPawns &&
                    fields[(rows[idx] - 2) * BOARD_SIZE + cols[idx] + 2] < 0)
                    return true;
            }
        }
    }
    return false;
}

__host__ __device__ void setAvailableFields(int row, int col, bool isWhite, bool* available, int* fields, int& numOfAvailable)
{
    bool shouldUpdateAvailable = available != nullptr;
    if (isWhite && row < BOARD_SIZE - 1)
    {
        if (col > 0 && fields[(row + 1) * BOARD_SIZE + col - 1] < 0)
        {
            if (shouldUpdateAvailable)
                available[(row + 1) * BOARD_SIZE + col - 1] = true;
            numOfAvailable++;
        }
        if (col < BOARD_SIZE - 1 && fields[(row + 1) * BOARD_SIZE + col + 1] < 0)
        {
            if (shouldUpdateAvailable)
                available[(row + 1) * BOARD_SIZE + col + 1] = true;
            numOfAvailable++;
        }
    }
    else if (!isWhite && row > 0)
    {
        if (col > 0 && fields[(row - 1) * BOARD_SIZE + col - 1] < 0)
        {
            if (shouldUpdateAvailable)
                available[(row - 1) * BOARD_SIZE + col - 1] = true;
            numOfAvailable++;
        }
        if (col < BOARD_SIZE - 1 && fields[(row - 1) * BOARD_SIZE + col + 1] < 0)
        {
            if (shouldUpdateAvailable)
                available[(row - 1) * BOARD_SIZE + col + 1] = true;
            numOfAvailable++;
        }
    }
}

__host__ __device__ void setAvailableQueenFields(int row, int col, bool* available, int* fields, int& numOfAvailable)
{
    bool shouldUpdateAvailable = available != nullptr;
    for (int r = row - 1, c = col - 1; r >= 0 && c >= 0; r--, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;
        if (shouldUpdateAvailable)
            available[r * BOARD_SIZE + c] = true;
        numOfAvailable++;
    }
    for (int r = row + 1, c = col + 1; r < BOARD_SIZE && c < BOARD_SIZE; r++, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;
        if (shouldUpdateAvailable)
            available[r * BOARD_SIZE + c] = true;
        numOfAvailable++;
    }
    for (int r = row - 1, c = col + 1; r >= 0 && c < BOARD_SIZE; r--, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;
        if (shouldUpdateAvailable)
            available[r * BOARD_SIZE + c] = true;
        numOfAvailable++;
    }
    for (int r = row + 1, c = col - 1; r < BOARD_SIZE && c >= 0; r++, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;
        if (shouldUpdateAvailable)
            available[r * BOARD_SIZE + c] = true;
        numOfAvailable++;
    }
}

__host__ __device__ void setAvailableKills(int* fields, int row, int col, int idx, bool isWhite, bool* available, int& numOfAvailable)
{
    int halfPawn = PAWN_ROWS * BOARD_SIZE / 2;
    if (isWhite)
    {
        if (col > 1 && row < BOARD_SIZE - 2 &&
            fields[(row + 2) * BOARD_SIZE + col - 2] < 0 &&
            fields[(row + 1) * BOARD_SIZE + col - 1] >= 0 &&
            fields[(row + 1) * BOARD_SIZE + col - 1] / halfPawn != idx / halfPawn)
        {
            available[(row + 2) * BOARD_SIZE + col - 2] = true;
            numOfAvailable++;
        }

        if (col < BOARD_SIZE - 2 && row < BOARD_SIZE - 2 &&
            fields[(row + 2) * BOARD_SIZE + col + 2] < 0 &&
            fields[(row + 1) * BOARD_SIZE + col + 1] >= 0 &&
            fields[(row + 1) * BOARD_SIZE + col + 1] / halfPawn != idx / halfPawn)
        {
            available[(row + 2) * BOARD_SIZE + col + 2] = true;
            numOfAvailable++;
        }
    }
    else if (!isWhite)
    {
        if (col > 1 && row > 1 &&
            fields[(row - 2) * BOARD_SIZE + col - 2] < 0 &&
            fields[(row - 1) * BOARD_SIZE + col - 1] >= 0 &&
            fields[(row - 1) * BOARD_SIZE + col - 1] / halfPawn != idx / halfPawn)
        {
            available[(row - 2) * BOARD_SIZE + col - 2] = true;
            numOfAvailable++;
        }
        if (col < BOARD_SIZE - 2 && row > 1 &&
            fields[(row - 2) * BOARD_SIZE + col + 2] < 0 &&
            fields[(row - 1) * BOARD_SIZE + col + 1] >= 0 &&
            fields[(row - 1) * BOARD_SIZE + col + 1] / halfPawn != idx / halfPawn)
        {
            available[(row - 2) * BOARD_SIZE + col + 2] = true;
            numOfAvailable++;
        }
    }
}

__host__ __device__ void setAvailableQueenKills(int* fields, int row, int col, int idx, bool* available, int& numOfAvailable)
{
    int halfPawn = PAWN_ROWS * BOARD_SIZE / 2;

    for (int r = row + 1, c = col - 1; r < BOARD_SIZE - 1 && c > 0; r++, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c - 1] < 0)
            {
                available[(r + 1) * BOARD_SIZE + c - 1] = true;
                numOfAvailable++;
            }
            break;
        }
    }
    for (int r = row + 1, c = col + 1; r < BOARD_SIZE - 1 && c < BOARD_SIZE - 1; r++, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c + 1] < 0)
            {
                available[(r + 1) * BOARD_SIZE + c + 1] = true;
                numOfAvailable++;
            }
            break;
        }
    }
    for (int r = row - 1, c = col - 1; r > 0 && c > 0; r--, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c - 1] < 0)
            {
                available[(r - 1) * BOARD_SIZE + c - 1] = true;
                numOfAvailable++;
            }
            break;
        }
    }
    for (int r = row - 1, c = col + 1; r > 0 && c < BOARD_SIZE - 1; r--, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c + 1] < 0)
            {
                available[(r - 1) * BOARD_SIZE + c + 1] = true;
                numOfAvailable++;
            }
            break;
        }
    }
}

__host__ void markAvailableFields(RectangleShape* fieldShapes, bool* available)
{
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        if (available[i])
            fieldShapes[i].setFillColor(Color::Color(125, 125, 125));
    }
}

__host__ __device__ void removePawn(int idx, int* rows, int* cols, int* fields)
{
    int targetIdx = fields[idx];
    rows[targetIdx] = -1;
    cols[targetIdx] = -1;
    fields[idx] = -1;
}

__host__ void handlePawnClick(int i, int* rows, int* cols, int* fields, RectangleShape* fieldShapes, bool* available, int& numOfAvailable,
    int& selectedPawnIdx, bool& performedOperation, bool isThereKill, bool* pawnHasKill, bool* isQueen, bool isChainKill = false)
{
    clearAvailableFields(available, numOfAvailable);
    if (isQueen[i])
    {
        if (isChainKill || pawnHasKill[i])
        {
            setAvailableQueenKills(fields, rows[i], cols[i], i, available, numOfAvailable);
        }
        else if (!isThereKill)
        {
            setAvailableQueenFields(rows[i], cols[i], available, fields, numOfAvailable);
        }
    }
    else
    {
        if (isChainKill)
        {
            setAvailableKills(fields, rows[i], cols[i], i, true, available, numOfAvailable);
            setAvailableKills(fields, rows[i], cols[i], i, false, available, numOfAvailable);
        }
        else if (!isThereKill)
        {
            setAvailableFields(rows[i], cols[i], i < (PAWN_ROWS* BOARD_SIZE / 2),
                available, fields, numOfAvailable);
        }
        else if (pawnHasKill[i])
        {
            setAvailableKills(fields, rows[i], cols[i], i, i < (PAWN_ROWS* BOARD_SIZE / 2), available, numOfAvailable);
        }
    }
    markAvailableFields(fieldShapes, available);
    selectedPawnIdx = i;
    performedOperation = true;
}

__host__ void markQueen(CircleShape* pawns, int idx)
{
    pawns[idx].setOutlineColor(Color::Yellow);
}

__host__ __device__ int trackPawnToRemove(int rowStart, int colStart, int rowEnd, int colEnd, int* fields)
{
    int diffR = rowEnd - rowStart > 0 ? 1 : -1;
    int diffC = colEnd - colStart > 0 ? 1 : -1;
    for (int r = rowStart + diffR, c = colStart + diffC; r != rowEnd; r += diffR, c += diffC)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            return (r * BOARD_SIZE + c);
    }
    return -1;
}

// from [start, end)
__host__ int h_getRandom(int start, int end)
{
    return rand() % (end - start) + start;
}

// from [start, end)
__device__ int d_getRandom(int start, int end, hiprandState* state)
{
    return end - ceilf(hiprand_uniform(state) * (end - start));
}

__host__ bool h_makeRandomAvailableMove(int* fields, int* rows, int* cols, bool* pawnHasKill, bool* isQueen, bool& blackTurn, bool* available, int& numOfWhite, int& numOfBlack, int pawnInChainKill = -1)
{
    bool isThereKill = false;
    int numOfPawnsWithKill = 0;
    int numOfAvailable = 0;
    clearAvailableFields(available, numOfAvailable);
    int targetPos = -1;
    int idx = blackTurn ? PAWN_ROWS * BOARD_SIZE / 2 - 1 : -1;
    if (pawnInChainKill >= 0)
    {
        isThereKill = true;

        idx = pawnInChainKill;
        if (isQueen[idx])
            setAvailableQueenKills(fields, rows[idx], cols[idx], idx, available, numOfAvailable);
        else
        {
            setAvailableKills(fields, rows[idx], cols[idx], idx, true, available, numOfAvailable);
            setAvailableKills(fields, rows[idx], cols[idx], idx, false, available, numOfAvailable);
        }
    }
    else {
        if (!blackTurn)
        {
            for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
            {
                pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                    cols[i], i) : hasKill(fields, i, rows, cols));
                if (pawnHasKill[i])
                {
                    isThereKill = true;
                    numOfPawnsWithKill++;
                }
            }
        }
        else
        {
            for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
            {
                pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                    cols[i], i) : hasKill(fields, i, rows, cols));
                if (pawnHasKill[i])
                {
                    isThereKill = true;
                    numOfPawnsWithKill++;
                }
            }
        }
        if (isThereKill)
        {
            int rndPawn = h_getRandom(0, numOfPawnsWithKill);
            int counter = -1;
            while (counter < rndPawn)
            {
                idx++;
                if (pawnHasKill[idx])
                {
                    counter++;
                }
            }
            if (isQueen[idx])
                setAvailableQueenKills(fields, rows[idx], cols[idx], idx, available, numOfAvailable);
            else
                setAvailableKills(fields, rows[idx], cols[idx], idx, idx < PAWN_ROWS* BOARD_SIZE / 2, available, numOfAvailable);
        }
        else
        {
            int numOfPossible = 0;
            int start = blackTurn ? PAWN_ROWS * BOARD_SIZE / 2 : 0;
            int end = blackTurn ? PAWN_ROWS * BOARD_SIZE : PAWN_ROWS * BOARD_SIZE / 2;
            for (int i = start; i < end; i++)
            {
                if (rows[i] >= 0)
                {
                    numOfAvailable = 0;
                    if (isQueen[i])
                    {
                        setAvailableQueenFields(rows[i], cols[i], nullptr, fields, numOfAvailable);
                    }
                    else
                    {
                        setAvailableFields(rows[i], cols[i], i < PAWN_ROWS* BOARD_SIZE / 2, nullptr, fields, numOfAvailable);
                    }
                    if (numOfAvailable > 0)
                    {
                        numOfPossible = numOfPossible + 1;
                        available[i] = true;
                    }
                }
            }
            // draw
            if (numOfPossible == 0)
            {
                numOfWhite = 0;
                numOfBlack = 0;
                return false;
            }
            int possibleIdx = h_getRandom(0, numOfPossible);
            int counter = 0;
            for (int i = start; i < end; i++)
            {
                if (available[i]) {
                    if (counter == possibleIdx)
                    {
                        idx = i;
                        break;
                    }
                    counter++;
                }
            }
            clearAvailableFields(available, numOfAvailable);
            if (isQueen[idx])
            {
                setAvailableQueenFields(rows[idx], cols[idx], available, fields, numOfAvailable);
            }
            else
            {
                setAvailableFields(rows[idx], cols[idx], idx < PAWN_ROWS* BOARD_SIZE / 2, available, fields, numOfAvailable);
            }
        }
    }
    int rndMove = h_getRandom(0, numOfAvailable);
    int avCounter = -1;

    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        if (available[i])
        {
            avCounter++;
            if (avCounter == rndMove)
            {
                targetPos = i;
                break;
            }
        }
    }
    fields[rows[idx] * BOARD_SIZE + cols[idx]] = -1;
    int pawnToRemove;
    if ((pawnToRemove = trackPawnToRemove(rows[idx], cols[idx], targetPos / BOARD_SIZE, targetPos % BOARD_SIZE, fields)) >= 0)
    {
        if (fields[pawnToRemove] < PAWN_ROWS * BOARD_SIZE / 2)
            numOfWhite--;
        else
            numOfBlack--;
        removePawn(pawnToRemove, rows, cols, fields);
    }
    fields[targetPos] = idx;
    rows[idx] = targetPos / BOARD_SIZE;
    cols[idx] = targetPos % BOARD_SIZE;

    bool blockChainKill = false;
    if ((idx >= PAWN_ROWS * BOARD_SIZE / 2
        && rows[idx] == 0) ||
        (idx < PAWN_ROWS * BOARD_SIZE / 2 &&
            rows[idx] == BOARD_SIZE - 1))
    {
        isQueen[idx] = true;
        blockChainKill = true;
    }
    int nextPawnInChainKill = -1;
    if (isThereKill && !blockChainKill && (isQueen[idx] ? hasQueenKill(fields, rows[idx],
        cols[idx], idx) : hasKill(fields, idx, rows, cols, true)))
    {
        nextPawnInChainKill = idx;
        return h_makeRandomAvailableMove(fields, rows, cols, pawnHasKill, isQueen, blackTurn, available, numOfWhite, numOfBlack, nextPawnInChainKill);
    }
    return numOfWhite > 0 && numOfBlack > 0;
}

__device__ bool d_makeRandomAvailableMove(int* fields, int* rows, int* cols, bool* pawnHasKill, bool* isQueen, bool& blackTurn, bool* available, int& numOfWhite, int& numOfBlack, hiprandState* state, int pawnInChainKill = -1)
{
    bool isThereKill = false;
    int numOfPawnsWithKill = 0;
    int numOfAvailable = 0;
    clearAvailableFields(available, numOfAvailable);
    int targetPos = -1;
    int idx = blackTurn ? PAWN_ROWS * BOARD_SIZE / 2 - 1 : -1;

    if (pawnInChainKill >= 0)
    {
        isThereKill = true;

        idx = pawnInChainKill;
        if (isQueen[idx])
            setAvailableQueenKills(fields, rows[idx], cols[idx], idx, available, numOfAvailable);
        else
        {
            setAvailableKills(fields, rows[idx], cols[idx], idx, true, available, numOfAvailable);
            setAvailableKills(fields, rows[idx], cols[idx], idx, false, available, numOfAvailable);
        }
    }
    else {
        if (!blackTurn)
        {
            for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
            {
                pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                    cols[i], i) : hasKill(fields, i, rows, cols));
                if (pawnHasKill[i])
                {
                    isThereKill = true;
                    numOfPawnsWithKill++;
                }
            }
        }
        else
        {
            for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
            {
                pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                    cols[i], i) : hasKill(fields, i, rows, cols));
                if (pawnHasKill[i])
                {
                    isThereKill = true;
                    numOfPawnsWithKill++;
                }
            }
        }
        if (isThereKill)
        {
            int rndPawn = d_getRandom(0, numOfPawnsWithKill, state);
            int counter = -1;
            while (counter < rndPawn)
            {
                idx++;
                if (pawnHasKill[idx])
                {
                    counter++;
                }
            }
            if (isQueen[idx])
                setAvailableQueenKills(fields, rows[idx], cols[idx], idx, available, numOfAvailable);
            else
                setAvailableKills(fields, rows[idx], cols[idx], idx, idx < PAWN_ROWS* BOARD_SIZE / 2, available, numOfAvailable);
        }
        else
        {
            int numOfPossible = 0;
            int start = blackTurn ? PAWN_ROWS * BOARD_SIZE / 2 : 0;
            int end = blackTurn ? PAWN_ROWS * BOARD_SIZE : PAWN_ROWS * BOARD_SIZE / 2;
            for (int i = start; i < end; i++)
            {
                if (rows[i] >= 0)
                {
                    numOfAvailable = 0;
                    if (isQueen[i])
                    {
                        setAvailableQueenFields(rows[i], cols[i], nullptr, fields, numOfAvailable);
                    }
                    else
                    {
                        setAvailableFields(rows[i], cols[i], i < PAWN_ROWS* BOARD_SIZE / 2, nullptr, fields, numOfAvailable);
                    }
                    if (numOfAvailable > 0)
                    {
                        numOfPossible = numOfPossible + 1;
                        available[i] = true;
                    }
                }
            }
            // draw
            if (numOfPossible == 0)
            {
                numOfWhite = 0;
                numOfBlack = 0;
                return false;
            }
            int possibleIdx = d_getRandom(0, numOfPossible, state);
            int counter = 0;
            for (int i = start; i < end; i++)
            {
                if (available[i]) {
                    if (counter == possibleIdx)
                    {
                        idx = i;
                        break;
                    }
                    counter++;
                }
            }
            clearAvailableFields(available, numOfAvailable);
            if (isQueen[idx])
            {
                setAvailableQueenFields(rows[idx], cols[idx], available, fields, numOfAvailable);
            }
            else
            {
                setAvailableFields(rows[idx], cols[idx], idx < PAWN_ROWS* BOARD_SIZE / 2, available, fields, numOfAvailable);
            }
        }
    }
    int rndMove = d_getRandom(0, numOfAvailable, state);
    int avCounter = -1;

    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        if (available[i])
        {
            avCounter++;
            if (avCounter == rndMove)
            {
                targetPos = i;
                break;
            }
        }
    }
    fields[rows[idx] * BOARD_SIZE + cols[idx]] = -1;
    int pawnToRemove;
    if ((pawnToRemove = trackPawnToRemove(rows[idx], cols[idx], targetPos / BOARD_SIZE, targetPos % BOARD_SIZE, fields)) >= 0)
    {
        if (fields[pawnToRemove] < PAWN_ROWS * BOARD_SIZE / 2)
            numOfWhite--;
        else
            numOfBlack--;
        removePawn(pawnToRemove, rows, cols, fields);
    }
    fields[targetPos] = idx;
    rows[idx] = targetPos / BOARD_SIZE;
    cols[idx] = targetPos % BOARD_SIZE;

    bool blockChainKill = false;
    if ((idx >= PAWN_ROWS * BOARD_SIZE / 2
        && rows[idx] == 0) ||
        (idx < PAWN_ROWS * BOARD_SIZE / 2 &&
            rows[idx] == BOARD_SIZE - 1))
    {
        isQueen[idx] = true;
        blockChainKill = true;
    }
    int nextPawnInChainKill = -1;
    if (isThereKill && !blockChainKill && (isQueen[idx] ? hasQueenKill(fields, rows[idx],
        cols[idx], idx) : hasKill(fields, idx, rows, cols, true)))
    {
        nextPawnInChainKill = idx;
        return d_makeRandomAvailableMove(fields, rows, cols, pawnHasKill, isQueen, blackTurn, available, numOfWhite, numOfBlack, state, nextPawnInChainKill);
    }
    return numOfWhite > 0 && numOfBlack > 0;
}

__host__ node* initNode(int* fields, int* rows, int* cols, bool* isQueen, bool blackTurn)
{
    node* state = new node;
    state->fields = new int[BOARD_SIZE * BOARD_SIZE];
    memcpy(state->fields, fields, BOARD_SIZE * BOARD_SIZE * sizeof(int));
    state->rows = new int[PAWN_ROWS * BOARD_SIZE];
    memcpy(state->rows, rows, PAWN_ROWS * BOARD_SIZE * sizeof(int));
    state->cols = new int[PAWN_ROWS * BOARD_SIZE];
    memcpy(state->cols, cols, PAWN_ROWS * BOARD_SIZE * sizeof(int));
    state->isQueen = new bool[PAWN_ROWS * BOARD_SIZE];
    memcpy(state->isQueen, isQueen, PAWN_ROWS * BOARD_SIZE * sizeof(bool));
    state->childs = nullptr;
    state->blackTurn = blackTurn;
    state->lastKill = -1;
    state->childSize = 0;
    state->avgReward = 0;
    state->howManyVisits = 0;

    return state;
}

__host__ void expandForPawnKills(int* fields, int row, int col, int idx, bool isWhite, node* root, bool changeTurn = true)
{
    int halfPawn = PAWN_ROWS * BOARD_SIZE / 2;
    if (isWhite)
    {
        if (col > 1 && row < BOARD_SIZE - 2 &&
            fields[(row + 2) * BOARD_SIZE + col - 2] < 0 &&
            fields[(row + 1) * BOARD_SIZE + col - 1] >= 0 &&
            fields[(row + 1) * BOARD_SIZE + col - 1] / halfPawn != idx / halfPawn)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

            child->rows[idx] = row + 2;
            child->cols[idx] = col - 2;

            removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row + 2) * BOARD_SIZE + col - 2] = idx;


            if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                child->lastKill = idx;

            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
            {
                if (!child->isQueen[idx])
                    child->lastKill = -1;
                child->isQueen[idx] = true;
            }

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }

        if (col < BOARD_SIZE - 2 && row < BOARD_SIZE - 2 &&
            fields[(row + 2) * BOARD_SIZE + col + 2] < 0 &&
            fields[(row + 1) * BOARD_SIZE + col + 1] >= 0 &&
            fields[(row + 1) * BOARD_SIZE + col + 1] / halfPawn != idx / halfPawn)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

            child->rows[idx] = row + 2;
            child->cols[idx] = col + 2;

            removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row + 2) * BOARD_SIZE + col + 2] = idx;

            if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                child->lastKill = idx;

            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
            {
                if (!child->isQueen[idx])
                    child->lastKill = -1;
                child->isQueen[idx] = true;
            }

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
    }
    else if (!isWhite)
    {
        if (col > 1 && row > 1 &&
            fields[(row - 2) * BOARD_SIZE + col - 2] < 0 &&
            fields[(row - 1) * BOARD_SIZE + col - 1] >= 0 &&
            fields[(row - 1) * BOARD_SIZE + col - 1] / halfPawn != idx / halfPawn)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

            child->rows[idx] = row - 2;
            child->cols[idx] = col - 2;

            removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row - 2) * BOARD_SIZE + col - 2] = idx;

            if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                child->lastKill = idx;

            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
            {
                if (!child->isQueen[idx])
                    child->lastKill = -1;
                child->isQueen[idx] = true;
            }

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
        if (col < BOARD_SIZE - 2 && row > 1 &&
            fields[(row - 2) * BOARD_SIZE + col + 2] < 0 &&
            fields[(row - 1) * BOARD_SIZE + col + 1] >= 0 &&
            fields[(row - 1) * BOARD_SIZE + col + 1] / halfPawn != idx / halfPawn)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

            child->rows[idx] = row - 2;
            child->cols[idx] = col + 2;

            removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row - 2) * BOARD_SIZE + col + 2] = idx;

            if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                child->lastKill = idx;

            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
            {
                if (!child->isQueen[idx])
                    child->lastKill = -1;
                child->isQueen[idx] = true;
            }

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
    }
}

__host__ void expandForQueenKill(int* fields, int row, int col, int idx, node* root, bool changeTurn = true)
{
    int halfPawn = PAWN_ROWS * BOARD_SIZE / 2;

    for (int r = row + 1, c = col - 1; r < BOARD_SIZE - 1 && c > 0; r++, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c - 1] < 0)
            {
                root->childSize = root->childSize + 1;
                node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
                if (newChilds == nullptr)
                {
                    return;
                }
                else root->childs = newChilds;

                node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

                child->rows[idx] = r + 1;
                child->cols[idx] = c - 1;

                removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

                child->fields[row * BOARD_SIZE + col] = -1;
                child->fields[(r + 1) * BOARD_SIZE + c - 1] = idx;

                if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                    child->lastKill = idx;

                root->childs[root->childSize - 1] = child;
                child->parent = root;
            }
            break;
        }
    }
    for (int r = row + 1, c = col + 1; r < BOARD_SIZE - 1 && c < BOARD_SIZE - 1; r++, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r + 1) * BOARD_SIZE + c + 1] < 0)
            {
                root->childSize = root->childSize + 1;
                node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
                if (newChilds == nullptr)
                {
                    return;
                }
                else root->childs = newChilds;

                node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

                child->rows[idx] = r + 1;
                child->cols[idx] = c + 1;

                removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

                child->fields[row * BOARD_SIZE + col] = -1;
                child->fields[(r + 1) * BOARD_SIZE + c + 1] = idx;

                if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                    child->lastKill = idx;

                root->childs[root->childSize - 1] = child;
                child->parent = root;
            }
            break;
        }
    }
    for (int r = row - 1, c = col - 1; r > 0 && c > 0; r--, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c - 1] < 0)
            {
                root->childSize = root->childSize + 1;
                node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
                if (newChilds == nullptr)
                {
                    return;
                }
                else root->childs = newChilds;

                node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

                child->rows[idx] = r - 1;
                child->cols[idx] = c - 1;

                removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

                child->fields[row * BOARD_SIZE + col] = -1;
                child->fields[(r - 1) * BOARD_SIZE + c - 1] = idx;

                if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                    child->lastKill = idx;

                root->childs[root->childSize - 1] = child;
                child->parent = root;
            }
            break;
        }
    }
    for (int r = row - 1, c = col + 1; r > 0 && c < BOARD_SIZE - 1; r--, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
        {
            if (fields[r * BOARD_SIZE + c] / halfPawn != idx / halfPawn
                && fields[(r - 1) * BOARD_SIZE + c + 1] < 0)
            {
                root->childSize = root->childSize + 1;
                node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
                if (newChilds == nullptr)
                {
                    return;
                }
                else root->childs = newChilds;

                node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, (changeTurn ? !(root->blackTurn) : root->blackTurn));

                child->rows[idx] = r - 1;
                child->cols[idx] = c + 1;

                removePawn(trackPawnToRemove(row, col, child->rows[idx], child->cols[idx], child->fields), child->rows, child->cols, child->fields);

                child->fields[row * BOARD_SIZE + col] = -1;
                child->fields[(r - 1) * BOARD_SIZE + c + 1] = idx;

                if ((child->isQueen[idx] ? hasQueenKill(child->fields, child->rows[idx], child->cols[idx], idx) : hasKill(child->fields, idx, child->rows, child->cols, true)))
                    child->lastKill = idx;

                root->childs[root->childSize - 1] = child;
                child->parent = root;
            }
            break;
        }
    }
}

__host__ void expandForPawnMoves(int row, int col, int idx, int* fields, node* root)
{
    if (!(root->blackTurn) && row < BOARD_SIZE - 1)
    {
        if (col > 0 && fields[(row + 1) * BOARD_SIZE + col - 1] < 0)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

            child->rows[idx] = row + 1;
            child->cols[idx] = col - 1;
            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row + 1) * BOARD_SIZE + col - 1] = idx;
            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
                child->isQueen[idx] = true;

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
        if (col < BOARD_SIZE - 1 && fields[(row + 1) * BOARD_SIZE + col + 1] < 0)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

            child->rows[idx] = row + 1;
            child->cols[idx] = col + 1;
            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row + 1) * BOARD_SIZE + col + 1] = idx;
            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
                child->isQueen[idx] = true;

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
    }
    else if (root->blackTurn && row > 0)
    {
        if (col > 0 && fields[(row - 1) * BOARD_SIZE + col - 1] < 0)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

            child->rows[idx] = row - 1;
            child->cols[idx] = col - 1;
            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row - 1) * BOARD_SIZE + col - 1] = idx;
            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
                child->isQueen[idx] = true;

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
        if (col < BOARD_SIZE - 1 && fields[(row - 1) * BOARD_SIZE + col + 1] < 0)
        {
            root->childSize = root->childSize + 1;
            node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
            if (newChilds == nullptr)
            {
                root->childSize = root->childSize - 1;
                return;
            }
            else root->childs = newChilds;

            node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

            child->rows[idx] = row - 1;
            child->cols[idx] = col + 1;
            child->fields[row * BOARD_SIZE + col] = -1;
            child->fields[(row - 1) * BOARD_SIZE + col + 1] = idx;
            if ((idx < PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == BOARD_SIZE - 1)
                || (idx >= PAWN_ROWS * BOARD_SIZE / 2 && child->rows[idx] == 0))
                child->isQueen[idx] = true;

            root->childs[root->childSize - 1] = child;
            child->parent = root;
        }
    }
}

__host__ void expandForQueenMoves(int row, int col, int idx, int* fields, node* root)
{
    for (int r = row - 1, c = col - 1; r >= 0 && c >= 0; r--, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;

        root->childSize = root->childSize + 1;
        node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
        if (newChilds == nullptr)
        {
            root->childSize = root->childSize - 1;
            return;
        }
        else root->childs = newChilds;

        node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

        child->rows[idx] = r;
        child->cols[idx] = c;
        child->fields[row * BOARD_SIZE + col] = -1;
        child->fields[r * BOARD_SIZE + c] = idx;

        root->childs[root->childSize - 1] = child;
        child->parent = root;
    }
    for (int r = row + 1, c = col + 1; r < BOARD_SIZE && c < BOARD_SIZE; r++, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;

        root->childSize = root->childSize + 1;
        node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
        if (newChilds == nullptr)
        {
            root->childSize = root->childSize - 1;
            return;
        }
        else root->childs = newChilds;

        node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

        child->rows[idx] = r;
        child->cols[idx] = c;
        child->fields[row * BOARD_SIZE + col] = -1;
        child->fields[r * BOARD_SIZE + c] = idx;

        root->childs[root->childSize - 1] = child;
        child->parent = root;
    }
    for (int r = row - 1, c = col + 1; r >= 0 && c < BOARD_SIZE; r--, c++)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;

        root->childSize = root->childSize + 1;
        node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
        if (newChilds == nullptr)
        {
            root->childSize = root->childSize - 1;
            return;
        }
        else root->childs = newChilds;

        node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

        child->rows[idx] = r;
        child->cols[idx] = c;
        child->fields[row * BOARD_SIZE + col] = -1;
        child->fields[r * BOARD_SIZE + c] = idx;

        root->childs[root->childSize - 1] = child;
        child->parent = root;
    }
    for (int r = row + 1, c = col - 1; r < BOARD_SIZE && c >= 0; r++, c--)
    {
        if (fields[r * BOARD_SIZE + c] >= 0)
            break;

        root->childSize = root->childSize + 1;
        node** newChilds = (node**)realloc(root->childs, root->childSize * sizeof(node*));
        if (newChilds == nullptr)
        {
            root->childSize = root->childSize - 1;
            return;
        }
        else root->childs = newChilds;

        node* child = initNode(root->fields, root->rows, root->cols, root->isQueen, !(root->blackTurn));

        child->rows[idx] = r;
        child->cols[idx] = c;
        child->fields[row * BOARD_SIZE + col] = -1;
        child->fields[r * BOARD_SIZE + c] = idx;

        root->childs[root->childSize - 1] = child;
        child->parent = root;
    }
}

__host__ void expandNode(node* root)
{
    if (root->lastKill >= 0)
    {

        if (root->isQueen[root->lastKill])
        {
            expandForQueenKill(root->fields, root->rows[root->lastKill], root->cols[root->lastKill], root->lastKill, root, false);
        }
        else
        {
            expandForPawnKills(root->fields, root->rows[root->lastKill], root->cols[root->lastKill], root->lastKill, true, root, false);
            expandForPawnKills(root->fields, root->rows[root->lastKill], root->cols[root->lastKill], root->lastKill, false, root, false);
        }
        return;

    }

    int start = root->blackTurn ? PAWN_ROWS * BOARD_SIZE / 2 : 0;
    int end = root->blackTurn ? PAWN_ROWS * BOARD_SIZE : PAWN_ROWS * BOARD_SIZE / 2;
    bool isThereKill = false;

    for (int i = start; i < end; i++)
    {
        if (root->rows[i] >= 0)
        {
            if (root->isQueen[i] && hasQueenKill(root->fields, root->rows[i], root->cols[i], i))
            {
                expandForQueenKill(root->fields, root->rows[i], root->cols[i], i, root);
                isThereKill = true;
            }
            else if (hasKill(root->fields, i, root->rows, root->cols))
            {
                expandForPawnKills(root->fields, root->rows[i], root->cols[i], i, !(root->blackTurn), root);
                isThereKill = true;
            }
        }

    }
    if (isThereKill) return;

    for (int i = start; i < end; i++)
    {
        if (root->rows[i] >= 0)
        {
            if (root->isQueen[i])
            {
                expandForQueenMoves(root->rows[i], root->cols[i], i, root->fields, root);
            }
            else
            {
                expandForPawnMoves(root->rows[i], root->cols[i], i, root->fields, root);
            }
        }
    }
}

__host__ float getUCBValue(node* parent, node* child)
{
    if (child->howManyVisits == 0) return INFINITY;
    return (float)(child->avgReward + 2 * sqrt(log(parent->howManyVisits) / (float)child->howManyVisits));
}

__host__ void freeNode(node* root)
{
    delete[] root->fields;
    delete[] root->rows;
    delete[] root->cols;
    delete[] root->isQueen;

    for (int i = 0; i < root->childSize; i++)
        delete root->childs[i];

    if (root->childs != nullptr)
        delete[] root->childs;
    delete root;
}

// inspired by wischk checkers program evalutaion function
// http://people.cs.uchicago.edu/~wiseman/checkers/
__host__ __device__ float evaluatePositionValue(int* rows, int* cols, bool* isQueen, bool blackEval)
{
    float bMaterialValue = 0;
    float wMaterialValue = 0;
    float tscore = 0;
    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
    {
        if (rows[i] >= 0)
        {
            if (isQueen[i]) wMaterialValue += QUEEN_VALUE;
            else wMaterialValue += PAWN_VALUE;

            if ((rows[i] == 3 && cols[i] == 3)
                || (rows[i] == 4 && cols[i] == 4)
                || (rows[i] == 5 && cols[i] == 3)
                || (rows[i] == 4 && cols[i] == 2))
                tscore -= PIECE_MIDDLE_CENTER;

            if ((rows[i] == 3 && cols[i] == 1)
                || (rows[i] == 4 && cols[i] == 0)
                || (rows[i] == 5 && cols[i] == 7)
                || (rows[i] == 4 && cols[i] == 6))
                tscore -= PIECE_MIDDLE_SIDE;

            if ((rows[i] == 0 && cols[i] == 0)
                || (rows[i] == 0 && cols[i] == 6))
                tscore -= PIECE_SIDE_GOALIES;


            if ((rows[i] == 0 && cols[i] == 2)
                || (rows[i] == 0 && cols[i] == 4))
                tscore -= PIECE_CENTER_GOALIES;

            if ((rows[i] == 0 && cols[i] == 6)
                || (rows[i] == 1 && cols[i] == 7))
                tscore -= PIECE_DOUBLE_CORNER;

            tscore -= rows[i] * PIECE_ROW_ADV;
        }
    }
    for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
    {
        if (rows[i] >= 0)
        {
            if (isQueen[i]) bMaterialValue += QUEEN_VALUE;
            else bMaterialValue += PAWN_VALUE;

            if ((rows[i] == 3 && cols[i] == 3)
                || (rows[i] == 4 && cols[i] == 4)
                || (rows[i] == 5 && cols[i] == 3)
                || (rows[i] == 4 && cols[i] == 2))
                tscore += PIECE_MIDDLE_CENTER;

            if ((rows[i] == 3 && cols[i] == 1)
                || (rows[i] == 4 && cols[i] == 0)
                || (rows[i] == 5 && cols[i] == 7)
                || (rows[i] == 4 && cols[i] == 6))
                tscore += PIECE_MIDDLE_SIDE;

            if ((rows[i] == 7 && cols[i] == 1)
                || (rows[i] == 7 && cols[i] == 7))
                tscore += PIECE_SIDE_GOALIES;

            if ((rows[i] == 7 && cols[i] == 3)
                || (rows[i] == 7 && cols[i] == 5))
                tscore += PIECE_CENTER_GOALIES;

            if ((rows[i] == 7 && cols[i] == 1)
                || (rows[i] == 6 && cols[i] == 0))
                tscore += PIECE_DOUBLE_CORNER;

            tscore += (7 - rows[i]) * PIECE_ROW_ADV;
        }
    }
    float maxMaterial = bMaterialValue > wMaterialValue ? bMaterialValue : wMaterialValue;
    float minMaterial = bMaterialValue < wMaterialValue ? bMaterialValue : wMaterialValue;
    tscore += (bMaterialValue - wMaterialValue) * maxMaterial / (minMaterial + 1);
    if (isnan(tscore))
    {
        tscore = 1;
    }
    return tscore * (blackEval ? 1 : -1);
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

void copyToFixedNode(node* root, fixedNode* fixed)
{
    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
    {
        fixed->cols[i] = root->cols[i];
        fixed->rows[i] = root->rows[i];
        fixed->isQueen[i] = root->isQueen[i];
    }
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        fixed->fields[i] = root->fields[i];
    }
}

template <unsigned int blockSize>
__global__ void d_runSimulation(fixedNode* root, float* rewards, bool blackTurn, int lastKill, bool blackEval, int g_numOfWhite, int g_numOfBlack)
{
    extern __shared__ volatile float sumRewards[MAX_BLOCK];
    unsigned int tid = threadIdx.x;
    int fields[BOARD_SIZE * BOARD_SIZE];
    int rows[PAWN_ROWS * BOARD_SIZE];
    int cols[PAWN_ROWS * BOARD_SIZE];
    bool isQueen[PAWN_ROWS * BOARD_SIZE];
    //int* fields = (int*)malloc( * sizeof(int));
    //if (fields == nullptr)
    //{
    //    return;
    //}
 /*   int* rows = (int*)malloc(PAWN_ROWS * BOARD_SIZE * sizeof(int));
    if (rows == nullptr)
    {
        free(fields);
        return;
    }
    int* cols = (int*)malloc(PAWN_ROWS * BOARD_SIZE * sizeof(int));
    if (cols == nullptr)
    {
        free(fields);
        free(rows);
        return;
    }
    bool* isQueen = (bool*)malloc(PAWN_ROWS * BOARD_SIZE * sizeof(bool));
    if (isQueen == nullptr)
    {
        free(fields);
        free(rows);
        free(cols);
        return;
    }*/


    hiprandState state;

    hiprand_init(clock64(), tid, 0, &state);


    int numOfWhite = g_numOfWhite, numOfBlack = g_numOfBlack;

    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
        fields[i] = root->fields[i];
    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
    {
        rows[i] = root->rows[i];
        cols[i] = root->cols[i];
        isQueen[i] = root->isQueen[i];
    }

    bool available[PAWN_ROWS * BOARD_SIZE];
    bool pawnHasKill[PAWN_ROWS * BOARD_SIZE];

    if (lastKill >= 0)
    {
        d_makeRandomAvailableMove(fields, rows, cols, pawnHasKill, isQueen,
            blackTurn, available, numOfWhite, numOfBlack, &state, lastKill);
    }
    for (int i = 0; i < MAX_MOVES; i++)
    {
        if (!d_makeRandomAvailableMove(fields, rows, cols, pawnHasKill, isQueen,
            blackTurn, available, numOfWhite, numOfBlack, &state)) break;
        blackTurn = !blackTurn;
    }
    sumRewards[tid] = evaluatePositionValue(rows, cols, isQueen, blackEval);
    __syncthreads();

    if (blockSize >= 512) { if (tid < 256) { sumRewards[tid] += sumRewards[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sumRewards[tid] += sumRewards[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sumRewards[tid] += sumRewards[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sumRewards, tid);
    if (tid == 0) rewards[blockIdx.x] = sumRewards[0];

}

bool d_initMemory(float** d_rewards, fixedNode** d_fixed, int blockNum)
{
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)d_rewards, blockNum * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return false;
    }
    cudaStatus = hipMalloc((void**)d_fixed, sizeof(fixedNode));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(d_rewards);
        return false;
    }

    return true;
}

void d_freeMemory(float* d_rewards, fixedNode* d_fixed)
{
    hipFree(d_rewards);
    hipFree(d_fixed);
}

bool deviceMakeEvaluation(node* root, bool blackEval, int player, float* d_rewards, fixedNode* d_fixed, std::chrono::nanoseconds* timeStamps)
{
    int numOfEvaluations = (player == PLAYER_ONE ? NUM_OF_EVAL_ONE : NUM_OF_EVAL_TWO);

    unsigned int blockSize = min(numOfEvaluations, 1024);
    unsigned int blockNum = (int)ceil(numOfEvaluations / (float)blockSize);

    hipError_t cudaStatus;

    fixedNode* h_fixed = (fixedNode*)malloc(sizeof(fixedNode));
    copyToFixedNode(root, h_fixed);

    auto gpuMemAllocStart1 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(d_fixed, h_fixed, sizeof(fixedNode), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return false;
    }
    auto gpuMemAllocEnd1 = std::chrono::high_resolution_clock::now();
    int baseNumOfWhite = 0;
    int baseNumOfBlack = 0;

    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
        if (root->rows[i] >= 0) baseNumOfWhite++;
    for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
        if (root->rows[i] >= 0) baseNumOfBlack++;

    auto deviceStart1 = std::chrono::high_resolution_clock::now();

    if (player == PLAYER_ONE)
    {
        d_runSimulation<BLOCK_SIZE_ONE> << < BLOCK_NUM_ONE_V, BLOCK_SIZE_ONE_V, MAX_BLOCK * sizeof(float) >> > (d_fixed, d_rewards, root->blackTurn, root->lastKill, blackEval, baseNumOfWhite, baseNumOfBlack);
    }
    else
    {
        d_runSimulation<BLOCK_SIZE_TWO> << <BLOCK_NUM_TWO_V, BLOCK_SIZE_TWO_V, MAX_BLOCK * sizeof(float) >> > (d_fixed, d_rewards, root->blackTurn, root->lastKill, blackEval, baseNumOfWhite, baseNumOfBlack);
    }

    cudaStatus = hipDeviceSynchronize();
    auto deviceEnd1 = std::chrono::high_resolution_clock::now();

    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize failed");
        return false;
    }

    float* rewards = (float*)malloc(blockNum * sizeof(float));
    auto gpuMemAllocStart2 = std::chrono::high_resolution_clock::now();
    cudaStatus = hipMemcpy(rewards, d_rewards, blockNum * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return false;
    }
    thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(d_rewards);
    auto gpuMemAllocEnd2 = std::chrono::high_resolution_clock::now();
    
    auto deviceStart2 = std::chrono::high_resolution_clock::now();
    float sumRewards = (float)thrust::reduce(dev_ptr, dev_ptr + blockNum, 0);
    auto deviceEnd2 = std::chrono::high_resolution_clock::now();
    
    root->avgReward = sumRewards / numOfEvaluations;

    timeStamps[0] += (deviceEnd1 - deviceStart1) + (deviceEnd2 - deviceStart2);
    timeStamps[1] += (gpuMemAllocEnd1 - gpuMemAllocStart1) + (gpuMemAllocEnd2 - gpuMemAllocStart2);

    return true;
}

void hostMakeEvaluation(node* root, bool blackEval, int player, std::chrono::nanoseconds* timeStamps)
{
    auto cpuStart = std::chrono::high_resolution_clock::now();
    float sumRewards = 0;
    int baseNumOfWhite = 0;
    int baseNumOfBlack = 0;

    int numOfWhite = 0, numOfBlack = 0;

    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
        if (root->rows[i] >= 0) baseNumOfWhite++;
    for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
        if (root->rows[i] >= 0) baseNumOfBlack++;

    for (int p = 0; p < (player == PLAYER_ONE ? NUM_OF_EVAL_ONE : NUM_OF_EVAL_TWO); p++)
    {
        node* copyNode = initNode(root->fields, root->rows, root->cols, root->isQueen, root->blackTurn);
        copyNode->lastKill = root->lastKill;
        bool* pawnHasKill = new bool[PAWN_ROWS * BOARD_SIZE];
        bool* available = new bool[BOARD_SIZE * BOARD_SIZE];
        bool blackTurn = copyNode->blackTurn;

        numOfWhite = baseNumOfWhite;
        numOfBlack = baseNumOfBlack;

        if (copyNode->lastKill >= 0)
        {
            h_makeRandomAvailableMove(copyNode->fields, copyNode->rows, copyNode->cols, pawnHasKill, copyNode->isQueen,
                blackTurn, available, numOfWhite, numOfBlack, copyNode->lastKill);
        }
        for (int i = 0; i < MAX_MOVES; i++)
        {
            if (!h_makeRandomAvailableMove(copyNode->fields, copyNode->rows, copyNode->cols, pawnHasKill, copyNode->isQueen,
                blackTurn, available, numOfWhite, numOfBlack)) break;
            blackTurn = !blackTurn;
        }
        if (numOfWhite != numOfBlack)
            sumRewards += evaluatePositionValue(copyNode->rows, copyNode->cols, copyNode->isQueen, blackEval);
        freeNode(copyNode);
        delete[] pawnHasKill;
        delete[] available;
    }
    root->avgReward = sumRewards / (float)(player == PLAYER_ONE ? NUM_OF_EVAL_ONE : NUM_OF_EVAL_TWO);
    auto cpuEnd = std::chrono::high_resolution_clock::now();
    timeStamps[2] += cpuEnd - cpuStart;
}

bool makeMCTSMove(int* fields, int* rows, int* cols, bool* isQueen, bool blackTurn, int player, std::chrono::nanoseconds* timeStamps)
{
    node* root = initNode(fields, rows, cols, isQueen, blackTurn);
    root->parent = nullptr;
    expandNode(root);

    if (root->childSize == 0)
        return false;


    timeStamps[0] = std::chrono::nanoseconds(0);
    timeStamps[1] = std::chrono::nanoseconds(0);
    timeStamps[2] = std::chrono::nanoseconds(0);

    auto gpuMemAllocStart = std::chrono::high_resolution_clock::now();
    float* d_rewards = nullptr;
    fixedNode* d_fixed;
    if (player == PLAYER_ONE && PARALLEL_PLAYER_ONE)
    {
        d_initMemory(&d_rewards, &d_fixed, BLOCK_NUM_ONE);
    }
    else if (player == PLAYER_TWO && PARALLEL_PLAYER_TWO)
    {
        d_initMemory(&d_rewards, &d_fixed, BLOCK_NUM_TWO);
    }
    auto gpuMemAllocEnd = std::chrono::high_resolution_clock::now();
    timeStamps[1] = gpuMemAllocEnd - gpuMemAllocStart;

    for (int p = 0; p < (player == PLAYER_ONE ? TREE_ITER_ONE : TREE_ITER_TWO); p++)
    {

        node* selectedChild = root;
        do {
            float maxUCB = getUCBValue(selectedChild, selectedChild->childs[0]);
            int idxWithBiggestUCB = 0;
            float handlerUCB = 0;
            for (int i = 1; i < selectedChild->childSize; i++)
                if ((handlerUCB = getUCBValue(selectedChild, selectedChild->childs[i])) > maxUCB)
                {
                    maxUCB = handlerUCB;
                    idxWithBiggestUCB = i;
                }
            selectedChild = selectedChild->childs[idxWithBiggestUCB];
        } while (selectedChild->childSize != 0);

        if (selectedChild->howManyVisits == 0)
        {
            if ((player == PLAYER_ONE && !PARALLEL_PLAYER_ONE) || (player == PLAYER_TWO && !PARALLEL_PLAYER_TWO))
                hostMakeEvaluation(selectedChild, blackTurn, player, timeStamps);
            else
                if (!deviceMakeEvaluation(selectedChild, blackTurn, player, d_rewards, d_fixed, timeStamps)) break;

            node* prev = selectedChild->parent;
            while (prev != nullptr)
            {
                prev->avgReward = 0;
                for (int i = 0; i < prev->childSize; i++)
                    prev->avgReward += prev->childs[i]->avgReward;
                prev->avgReward /= prev->childSize;
                prev->howManyVisits = prev->howManyVisits + 1;
                prev = prev->parent;
            }
        }
        else
        {
            expandNode(selectedChild);
        }
        selectedChild->howManyVisits = selectedChild->howManyVisits + 1;
    }

    node* selectedMove = root;
    do {
        float resultReward = selectedMove->childs[0]->avgReward;
        float resultHandler = 0;
        int resultIdx = 0;

        for (int i = 1; i < selectedMove->childSize; i++)
            if ((resultHandler = selectedMove->childs[i]->avgReward) > resultReward)
            {
                resultReward = resultHandler;
                resultIdx = i;
            }

        selectedMove = selectedMove->childs[resultIdx];
    } while (selectedMove->lastKill >= 0 && selectedMove->childSize > 0);

    // this shouldn't ever happen if tree is at least with few levels
    if (selectedMove->lastKill >= 0)
    {
        bool* pawnHasKill = new bool[PAWN_ROWS * BOARD_SIZE];
        bool* available = new bool[BOARD_SIZE * BOARD_SIZE];
        bool* isQueen = new bool[PAWN_ROWS * BOARD_SIZE];
        int numOfWhite = 0, numOfBlack = 0;
        for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
            if (selectedMove->rows[i] >= 0) numOfWhite++;
        for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
            if (selectedMove->rows[i] >= 0) numOfBlack++;
        h_makeRandomAvailableMove(selectedMove->fields, selectedMove->rows, selectedMove->cols, pawnHasKill, isQueen, selectedMove->blackTurn, available, numOfWhite, numOfBlack, selectedMove->lastKill);
        delete[] pawnHasKill;
        delete[] available;
        delete[] isQueen;
    }
    for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
    {
        fields[i] = selectedMove->fields[i];
    }
    for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
    {
        rows[i] = selectedMove->rows[i];
        cols[i] = selectedMove->cols[i];
        isQueen[i] = selectedMove->isQueen[i];
    }
    d_freeMemory(d_rewards, d_fixed);
    freeNode(root);

    return true;
}

void printOutTimes(std::chrono::nanoseconds* timeStamps, int blackTurn)
{
    string outputFile = "output.txt";
    ofstream output;
    output.open(outputFile, ios::app);

    auto deviceTime = std::chrono::duration_cast<std::chrono::microseconds>(timeStamps[0]).count();
    auto deviceMemoryTime = std::chrono::duration_cast<std::chrono::microseconds>(timeStamps[1]).count();
    auto cpuTime = std::chrono::duration_cast<std::chrono::microseconds>(timeStamps[2]).count();

    output << blackTurn << " " << MAX_MOVES << " "
        << TREE_ITER_ONE << " " << TREE_ITER_TWO << " "
        << NUM_OF_EVAL_ONE << " " << NUM_OF_EVAL_TWO << " "
        << deviceTime << " " << deviceMemoryTime << " " << cpuTime << endl;
    output.close();
}

int main()
{
    RenderWindow window{ VideoMode(WINDOW_WIDTH, WINDOW_HEIGHT), "Checkers" };
    RectangleShape* fieldShapes = new RectangleShape[BOARD_SIZE * BOARD_SIZE];
    CircleShape* pawns = new CircleShape[PAWN_ROWS * BOARD_SIZE];
    int* fields = new int[BOARD_SIZE * BOARD_SIZE];
    int* rows = new int[PAWN_ROWS * BOARD_SIZE];
    int* cols = new int[PAWN_ROWS * BOARD_SIZE];
    bool* pawnHasKill = new bool[PAWN_ROWS * BOARD_SIZE];
    bool* available = new bool[BOARD_SIZE * BOARD_SIZE];
    bool* isQueen = new bool[PAWN_ROWS * BOARD_SIZE];
    int selectedPawnIdx = -1;
    bool performedOperation = false;
    bool blackTurn = false;
    int pawnInChainKill = -1;
    int pawnToRemove = -1;
    bool blockChainKill = false;
    setupFields(fieldShapes, fields);
    setupPawns(pawns, fields, rows, cols, pawnHasKill, isQueen);
    unsigned t = time(NULL);
    srand(t);
    std::chrono::nanoseconds timeStamps[3];
    // 0 is for device time
    // 1 is for device memory operations
    // 2 is for cpu time

    window.setFramerateLimit(25);
    Event event;
    int numOfAvailable = 0;
    bool isThereKill = false;

    while (true)
    {
        window.clear(Color::Black);
        window.pollEvent(event);
        if (event.type == Event::Closed)
        {
            window.close();
            break;
        }

        if (PLAYER_VS_AI == 1)
        {
            if (blackTurn)
            {

                if (!makeMCTSMove(fields, rows, cols, isQueen, blackTurn, PLAYER_TWO, timeStamps)) break;
                printOutTimes(timeStamps, blackTurn);
                blackTurn = !blackTurn;
                for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
                {
                    if (rows[i] >= 0)
                    {
                        setPawnPosition(pawns[i], rows[i], cols[i]);
                        pawns[i].setRadius(PAWN_SIZE / 2);
                    }
                    else
                        pawns[i].setRadius(0);
                    if (isQueen[i]) markQueen(pawns, i);
                }
                isThereKill = false;
                for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
                {
                    pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                        cols[i], i) : hasKill(fields, i, rows, cols));
                    if (pawnHasKill[i])
                    {
                        isThereKill = true;
                    }
                }
            }
            else if (event.type == Event::MouseButtonPressed)
            {

                if (!blackTurn)
                {
                    performedOperation = false;
                    recolorFields(fieldShapes);
                    Vector2f mousePosition = (Vector2f)Mouse::getPosition(window);

                    if (pawnInChainKill >= 0)
                    {
                        if (isClickInShape(pawns[pawnInChainKill], mousePosition))
                        {
                            handlePawnClick(pawnInChainKill, rows, cols, fields, fieldShapes, available, numOfAvailable, selectedPawnIdx,
                                performedOperation, isThereKill, pawnHasKill, isQueen, true);

                        }
                    }
                    else {
                        if (!blackTurn)
                            for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
                            {
                                if (isClickInShape(pawns[i], mousePosition))
                                {
                                    handlePawnClick(i, rows, cols, fields, fieldShapes, available, numOfAvailable, selectedPawnIdx, performedOperation, isThereKill, pawnHasKill, isQueen);
                                    break;
                                }
                            }
                        else
                            for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
                            {
                                if (isClickInShape(pawns[i], mousePosition))
                                {
                                    handlePawnClick(i, rows, cols, fields, fieldShapes, available, numOfAvailable, selectedPawnIdx, performedOperation, isThereKill, pawnHasKill, isQueen);
                                    break;
                                }
                            }
                    }
                    if (!performedOperation && selectedPawnIdx >= 0)
                    {
                        for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
                        {
                            if (available[i] && isClickInShape(fieldShapes[i], mousePosition))
                            {
                                setPawnPosition(pawns[selectedPawnIdx], i / BOARD_SIZE, i % BOARD_SIZE);
                                if ((pawnToRemove = trackPawnToRemove(rows[selectedPawnIdx], cols[selectedPawnIdx], i / BOARD_SIZE, i % BOARD_SIZE, fields)) >= 0)
                                {
                                    pawns[fields[pawnToRemove]].setRadius(0);
                                    removePawn(pawnToRemove, rows, cols, fields);
                                }
                                fields[rows[selectedPawnIdx] * BOARD_SIZE + cols[selectedPawnIdx]] = -1;
                                fields[i] = selectedPawnIdx;
                                rows[selectedPawnIdx] = i / BOARD_SIZE;
                                cols[selectedPawnIdx] = i % BOARD_SIZE;
                                blockChainKill = false;
                                if ((selectedPawnIdx >= PAWN_ROWS * BOARD_SIZE / 2
                                    && rows[selectedPawnIdx] == 0) ||
                                    (selectedPawnIdx < PAWN_ROWS * BOARD_SIZE / 2 &&
                                        rows[selectedPawnIdx] == BOARD_SIZE - 1))
                                {
                                    markQueen(pawns, selectedPawnIdx);
                                    isQueen[selectedPawnIdx] = true;
                                    blockChainKill = true;
                                }

                                clearAvailableFields(available, numOfAvailable);
                                pawnInChainKill = -1;
                                if (isThereKill && !blockChainKill && (isQueen[selectedPawnIdx] ? hasQueenKill(fields, rows[selectedPawnIdx],
                                    cols[selectedPawnIdx], selectedPawnIdx) : hasKill(fields, selectedPawnIdx, rows, cols, true)))
                                {
                                    pawnInChainKill = selectedPawnIdx;
                                }
                                else
                                {
                                    blackTurn = !blackTurn;
                                    isThereKill = false;
                                    if (!blackTurn)
                                    {
                                        for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
                                        {
                                            pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                                                cols[i], i) : hasKill(fields, i, rows, cols));
                                            if (pawnHasKill[i])
                                                isThereKill = true;
                                        }
                                    }
                                    else
                                    {
                                        for (int i = PAWN_ROWS * BOARD_SIZE / 2; i < PAWN_ROWS * BOARD_SIZE; i++)
                                        {
                                            pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                                                cols[i], i) : hasKill(fields, i, rows, cols));
                                            if (pawnHasKill[i])
                                                isThereKill = true;
                                        }
                                    }
                                }
                                selectedPawnIdx = -1;
                                break;
                            }
                        }
                    }
                }
                event.type = Event::MouseButtonReleased;
            }
        }
        else if (PLAYER_VS_AI == 0)
        {
            if (!makeMCTSMove(fields, rows, cols, isQueen, blackTurn, blackTurn ? PLAYER_TWO : PLAYER_ONE, timeStamps)) break;
            printOutTimes(timeStamps, blackTurn);
            Time t = sf::seconds(1);
            sleep(t);
            blackTurn = !blackTurn;
            for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
            {
                if (rows[i] >= 0)
                {
                    setPawnPosition(pawns[i], rows[i], cols[i]);
                    pawns[i].setRadius(PAWN_SIZE / 2);
                }
                else
                    pawns[i].setRadius(0);
                if (isQueen[i]) markQueen(pawns, i);
            }
            isThereKill = false;
            for (int i = 0; i < PAWN_ROWS * BOARD_SIZE / 2; i++)
            {
                pawnHasKill[i] = rows[i] >= 0 && (isQueen[i] ? hasQueenKill(fields, rows[i],
                    cols[i], i) : hasKill(fields, i, rows, cols));
                if (pawnHasKill[i])
                {
                    isThereKill = true;
                }
            }
        }

        for (int i = 0; i < BOARD_SIZE * BOARD_SIZE; i++)
            window.draw(fieldShapes[i]);
        for (int i = 0; i < PAWN_ROWS * BOARD_SIZE; i++)
            if (rows[i] >= 0) window.draw(pawns[i]);

        window.display();
    }
    delete[] fields;
    delete[] pawns;
    return 0;
}

